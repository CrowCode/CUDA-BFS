
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>

#define NUM_THREADS 1000
#define BLOCK_WIDTH 1000
#define N_VERTEX 1024  //gives N_VERTEX*(N_VERTEX-1) edges in case of non-directed graph
#define INF INT_MAX
#define N_R 800

__global__ void bfs_kernel(int* d_V, int* d_E, int* d_F, int* d_X, int* d_C) {

	int i = 0;
	int currentIndex = 0;
	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	int firstIndex = myId * N_VERTEX;
	int lastIndex = firstIndex + N_VERTEX;

	if (d_F[myId]) {
		d_F[myId] = 0;
		d_X[myId] = 1;
		for (i = firstIndex; i < lastIndex; i++) {
		//Consider only neighbors
			if (d_E[i] == 1) { //this means that vertex (i/N_VERTEX) and (i%N_VERTEX) are connected
				currentIndex = i % N_VERTEX;
				if (d_X[currentIndex] == 0) {    //if it is not visited
					d_C[currentIndex] = d_C[myId] + 1;
					d_F[currentIndex] = 1;
				}
			}
		}
	}
}


//This function returns 0 if the given matrix has only zeros and an integer if not.
int isFEmpty(int* h_F) {

	int i = 0;
	int result = 0;

	for (i = 0; i < N_VERTEX; i++) 
	{
		result += h_F[i];
	}
	return result;
}

void printArray(int* array,int nCol, int arraySize) {
	int i = 0;

	printf("-------------------------------------------------\n");
	for (i = 0; i < arraySize; i++) {
		if(i%nCol==0) printf("\n");
		printf("[%d] ", array[i]);
	}
	printf("-------------------------------------------------\n");
}

int main(int argc, char *argv[]) {

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);
	int counter = 0;
	float millisecond = 0;
	int is_F_Empty = 1;
	int i = 0, j = 0;
	int nEdges = N_VERTEX * N_VERTEX; //this is not exactly the max number of the edges, but the size
	//of a matrix describing connectivity of each pair of vertices

	int* d_V;
	int* d_E;
	int* d_F;
	int* d_X;
	int* d_C;

	int h_V[N_VERTEX];      //Vertex
	int h_E[nEdges];        //Edge
	int h_F[N_VERTEX];      //Frontier
	int h_X[N_VERTEX];      //Visited
	int h_C[N_VERTEX];      //Cost (Distance from the beginning vertex)
    /*********************************/
   
    /*****************    EXAMPLE I ****************/
	/*
	h_E[0] = 0;
	h_E[1] = 0;
	h_E[2] = 0;
	h_E[3] = 1;
	h_E[4] = 1;
	h_E[5] = 0;

	h_E[6] = 0;
	h_E[7] = 0;
	h_E[8] = 1;
	h_E[9] = 1;
	h_E[10] = 1;
	h_E[11] = 0;

	h_E[12] = 0;
	h_E[13] = 1;
	h_E[14] = 0;
	h_E[15] = 0;
	h_E[16] = 0;
	h_E[17] = 0;

	h_E[18] = 1;
	h_E[19] = 1;
	h_E[20] = 0;
	h_E[21] = 0;
	h_E[22] = 0;
	h_E[23] = 1;

	h_E[24] = 1;
	h_E[25] = 1;
	h_E[26] = 0;
	h_E[27] = 0;
	h_E[28] = 0;
	h_E[29] = 0;

	h_E[30] = 0;
	h_E[31] = 0;
	h_E[32] = 0;
	h_E[33] = 1;
	h_E[34] = 0;
	h_E[35] = 0;
	*/
/*******************END I*********************/
   
/*****************    EXAMPLE II ***************/
	/*	
	for (i=0; i < nEdges; i++)
	{
		if(i/N_VERTEX == (i%N_VERTEX + 1))
			h_E[i] = 1;
		else
			h_E[i] = 0;

	}
	h_E[N_VERTEX - 1] = 1;
	*/
/*******************END II********************/

/******************* EXAMPLE III *************/
	for (i=0; i < nEdges; i++){
		if(2*(i/N_VERTEX)+1 == (i%N_VERTEX))
			h_E[i] = 1;
		else if(2*(i/N_VERTEX)+2 == (i%N_VERTEX))
			h_E[i] = 1;
		else
			h_E[i] = 0;
		
	}
	for(j=0; j< N_R; j++){
		i = rand() % nEdges;
		if(h_E[i] != 1)
			h_E[i]=1;
	}
//	printArray(h_E,N_VERTEX, nEdges);

/******************* END III ****************/
	for (i = 0; i < N_VERTEX; i++) 
	{
		h_V[i] = i;     //index
		h_F[i] = 0;     //false
		h_X[i] = 0;     //false
		h_C[i] = INF;   //infinity
	} 
/*********************TEST**********************/

    h_F[0] = 1;   //true
    h_C[0] = 0;

	hipMalloc((void**) &d_V, sizeof(int) * N_VERTEX);
	hipMalloc((void**) &d_E, sizeof(int) * nEdges);
	hipMalloc((void**) &d_F, sizeof(int) * N_VERTEX);
	hipMalloc((void**) &d_X, sizeof(int) * N_VERTEX);
	hipMalloc((void**) &d_C, sizeof(int) * N_VERTEX);

	hipMemset((void*) &d_V, 0, sizeof(int) * N_VERTEX);
	hipMemset((void*) &d_F, 0, sizeof(int) * N_VERTEX);
	hipMemset((void*) &d_X, 0, sizeof(int) * N_VERTEX);
	hipMemset((void*) &d_C, 0, sizeof(int) * N_VERTEX);

	hipMemcpy(d_V, h_V, sizeof(int) * N_VERTEX, hipMemcpyHostToDevice);
	hipMemcpy(d_E, h_E, sizeof(int) * nEdges, hipMemcpyHostToDevice);
	hipEventRecord(begin);
	while (is_F_Empty) 
	{//while h_F is not all zeros
		hipMemcpy(d_F, h_F, sizeof(int) * N_VERTEX, hipMemcpyHostToDevice);
		hipMemcpy(d_X, h_X, sizeof(int) * N_VERTEX, hipMemcpyHostToDevice);
		hipMemcpy(d_C, h_C, sizeof(int) * N_VERTEX, hipMemcpyHostToDevice);

		bfs_kernel<<<2,512>>>(d_V, d_E, d_F, d_X, d_C);

		hipMemcpy(h_F, d_F, sizeof(int) * N_VERTEX, hipMemcpyDeviceToHost);
		hipMemcpy(h_X, d_X, sizeof(int) * N_VERTEX, hipMemcpyDeviceToHost);
		hipMemcpy(h_C, d_C, sizeof(int) * N_VERTEX, hipMemcpyDeviceToHost);

		is_F_Empty = isFEmpty(h_F);
        
		printf("_________________________\n");
		printf("\nLOOP COUNTER: %d\n", counter);
		counter++;
	}
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&millisecond, begin, end);
	printf("Second: %f", millisecond);
	printArray(h_C, N_VERTEX, N_VERTEX);
	hipFree(d_F);
	hipFree(d_X);
	hipFree(d_C);
	hipFree(d_V);
	hipFree(d_E);

	return EXIT_SUCCESS;
}
